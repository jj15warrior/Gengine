#include "hip/hip_runtime.h"
#include <iostream>
#include <GL/glut.h>
#include <GL/gl.h>
#include <GL/glu.h>
#include <math.h>
#include <chrono>
#include <thread>

#define DEBUG 1

float deltaTime = 0.0;

using namespace std;

void calcDeltaTime(){
    static auto lastTime = chrono::high_resolution_clock::now();
    auto currentTime = chrono::high_resolution_clock::now();
    deltaTime = chrono::duration_cast<chrono::microseconds>(currentTime - lastTime).count() / 1000000.0;
    lastTime = currentTime;
    cout << "fps: " << 1.0/deltaTime << endl;
}
struct color{
    float r, g, b;
    color(float r, float g, float b){
        this->r = r;
        this->g = g;
        this->b = b;
    }
};
class HelperFunctions{
    public:
        static void drawCircle(float x, float y, float radius, color c = color(1.0, 1.0, 1.0)){
            glBegin(GL_POLYGON);
            for(int i = 0; i < 360; i++){
                float degInRad = i * M_PI / 180;
                glColor3f(c.r, c.g, c.b);
                glVertex2f(cos(degInRad) * radius + x, sin(degInRad) * radius + y);
            }
            glEnd();
        }

};
void resize(int width, int height) {
    if (width > height){
        glViewport((width - height) / 2, 0, height, height);
    }
    else{
        glViewport(0, (height-width)/2, width, width);
    }
}
void display(){
    glClear(GL_COLOR_BUFFER_BIT);
    glClearColor(0.0, 0.0, 0.0, 1.0);
    glColor3f(0.0, 1.0, 0.0);
    HelperFunctions h = HelperFunctions();
    h.drawCircle(0.0, 0.0, 0.5, color(1.0, 0.0, 0.0));
    glFlush();

    calcDeltaTime();
    glutPostRedisplay();
}

void mouse_move(int x, int y){
    cout << "Mouse move: " << x << ", " << y << endl;
}

int main(int argc, char** argv){
    glutInit(&argc,argv);
    glutInitDisplayMode(GLUT_SINGLE | GLUT_RGB);
    glutInitWindowSize(1600, 900);
    glutCreateWindow("OpenGL - First window demo");
    glutDisplayFunc(display);
    glutPassiveMotionFunc(mouse_move);
    glutReshapeFunc(resize);
    glutMainLoop();
    return 0;
}
